#include "hip/hip_runtime.h"
//���������� ������������� ��������� ��������-���������� 
//��������� � �������������� CUDA

#include <stdio.h>
#include <stdlib.h>
#include <conio.h>
#include <windows.h>
#include <time.h>
#include <math.h>

#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"
#include "hip/hip_runtime.h"
#include ""


#define SIGMA      (hipfftDoubleReal(0.1))
#define SIGMA2     (SIGMA  * SIGMA)
#define SIGMA4     (SIGMA2 * SIGMA2)

__global__ void set_rhs(int n, hipfftDoubleReal* rhs) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;

	if (i < n && j < n) {
		hipfftDoubleReal h = 1 / hipfftDoubleReal(n);
		hipfftDoubleReal x = i * h - hipfftDoubleReal(0.5);
		hipfftDoubleReal y = j * h - hipfftDoubleReal(0.5);
		hipfftDoubleReal s = x * x + y * y;

		rhs[j * n + i] = (s - 2 * SIGMA2) * exp(-s / (2 * SIGMA2)) / SIGMA4;
	}
}

__global__ void solve_transformed(int n,
	const hipfftDoubleComplex* rhs, hipfftDoubleComplex* u) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;

	int m = n / 2 + 1;
	if (i < m && j < n) {
		hipfftDoubleComplex t = rhs[j * m + i];
		const double M_PI = 3.14;
		hipfftDoubleReal w = hipfftDoubleReal(M_PI) * (i < n / 2 ? i : i - n);
		hipfftDoubleReal v = hipfftDoubleReal(M_PI) * (j < n / 2 ? j : j - n);
		hipfftDoubleReal s = (!i && !j) ? 1 : -4 * (w * w + v * v);

		t.x /= s;
		t.y /= s;

		u[j * m + i] = t;
	}
}

__global__ void scale_and_shift(int n, hipfftDoubleReal* u, double shift) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;

	if (i < n && j < n)
		u[j * n + i] = (u[j * n + i] - shift) / (n * n);
}

extern "C" int fft_gpu(int n, double* u) {
	dim3 blk(32, 2);
	dim3 rgrd((n + blk.x - 1) / blk.x, (n + blk.y - 1) / blk.y);
	dim3 cgrd((n / 2 + blk.x) / blk.x, (n + blk.y - 1) / blk.y);
	double shift;

	hipfftDoubleComplex* v = NULL;
	hipError_t cuerr = hipMalloc((void**)&v, n * (n / 2 + 1) * sizeof(hipfftDoubleComplex));
	if (cuerr != hipSuccess)
	{
		fprintf(stderr, "Cannot create gpu memory buffer for v: %s\n",
			hipGetErrorString(cuerr));
		return 1;
	}

	set_rhs << <rgrd, blk >> >(n, u);

	hipfftHandle forward, inverse;
	hipfftPlan2d(&forward, n, n, HIPFFT_D2Z);
	hipfftPlan2d(&inverse, n, n, HIPFFT_Z2D);
	hipfftExecD2Z(forward, u, v);

	solve_transformed << <cgrd, blk >> >(n, v, v);
	hipfftExecZ2D(inverse, v, u);
	hipMemcpy(&shift, u, sizeof(double), hipMemcpyDeviceToHost);
	scale_and_shift << <rgrd, blk >> >(n, u, shift);
	hipFree(v);

	hipfftDestroy(forward);
	hipfftDestroy(inverse);

	return 0;
}